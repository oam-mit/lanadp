#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "dcmtk/dcmimgle/dcmimage.h"


__constant__ int powers[9]={0,1,2,4,8,16,32,64,128};

__device__ int getGlobalID(int row,int col,int cols){
    return row*cols+col;
}

__device__ int getCircularIndex(int i){
    
    if (i<1) {
        return 8 + i;
    }
    else if (i>8 )
      return (i%9 )+1;
    else 
      return i;
  
}

//dichom, dcmtk
__global__ void LANADP (unsigned int *arr, unsigned char *out,unsigned int rows,unsigned int cols) {
    
   // printf("Yes");
    int global_row = threadIdx.y + blockDim.y * blockIdx.y;
    int global_col = threadIdx.x + blockDim.x * blockIdx.x;

    if(global_row <rows && global_col<cols){
         if(global_row == 0 ||global_col ==0 || global_col==cols-1 || global_row==rows-1 ){
            //    if(global_row==0 && global_col==0){
            //         for(int i=0;i<3;i++){
            //             for(int i1=0;i1<3;i1++){
            //                 printf("%d  ",arr[i*cols+i1]);
            //             }

            //             printf("\n");
            //         }
            //    }
	  // printf("ID: %d\n",getGlobalID(global_row,global_col+1,cols));
        }

          else {
              
              int mapping[9];
              mapping[1]=arr[getGlobalID(global_row,global_col+1,cols)];
              mapping[2]=arr[getGlobalID(global_row-1,global_col+1,cols)];
              mapping[3]=arr[getGlobalID(global_row-1,global_col,cols)];
              mapping[4]=arr[getGlobalID(global_row-1,global_col-1,cols)];
              mapping[5]=arr[getGlobalID(global_row,global_col-1,cols)];
              mapping[6]=arr[getGlobalID(global_row+1,global_col-1,cols)];
              mapping[7]=arr[getGlobalID(global_row+1,global_col,cols)];
              mapping[8]=arr[getGlobalID(global_row+1,global_col+1,cols)];


              int ans=0;
              for (int i=1;i<=8;i++) {
                  
                  float avg1 = ((float)(mapping[getCircularIndex(i+1)] + mapping[getCircularIndex(i+2)]))/2.0;
                  float avg2 = ((float)(mapping[getCircularIndex(i-1)] + mapping[getCircularIndex(i-2)]))/2.0;

                  //printf("Number: %d, Avg1: %f, Avg2: %f\n",mapping[i],avg1,avg2);

                  if (avg1>=(float)mapping[i] && avg2>=(float)mapping[i] || (avg1<=(float)mapping[i] && avg2<=(float)mapping[i]) ){
                      
                      //printf("Adding:%d\n",powers[i]);
                      ans+=powers[i];
                  }
              }

              

               out[(global_row-1)*(cols-2)+(global_col-1)] = ans;


              
            
                
        }
    
    }

   
  
  
}


int main(int argc, char *argv[]) {

    int row,column;
    
    DicomImage *img=new DicomImage(argv[1]);
    

    if(img != NULL && img->getStatus()==EIS_Normal) {

        if(img->isMonochrome()) {

            img->setMinMaxWindow();
			
			row=img->getHeight();
			column=img->getWidth();

           

            unsigned int *arr = new unsigned int [row*column];

            printf("Rows: %d\tColumns:%d\n",row,column);

            img->getOutputData(arr,row*column*sizeof(unsigned int));

            unsigned char output[row-2][column-2];

            int output_rows=row-2,output_colums=column-2;

            int size_arr = row*column*sizeof(unsigned int);
            int size_out = output_rows*output_colums*sizeof(unsigned char);

            //memory variables
            unsigned int *d_arr; 
            unsigned char *d_output;

            //allocating memory
            hipMalloc((void **)&d_arr,size_arr);
            hipMalloc((void **)&d_output,size_out);
        
            //copying data
            hipMemcpy(d_arr,arr,size_arr,hipMemcpyHostToDevice); 

	//     float no_threads;
	// 	printf("Enter threads\n");
	// scanf("%f",&no_threads);


            dim3 numberOfBlocks(ceil((float)row/16.0),ceil((float)column/16.0),1);
            dim3 numberOfThreads(16,16,1);

	// printf("\n%d %d %d",numberOfBlocks.x,numberOfBlocks.y,numberOfBlocks.z);
	// printf("\n%d %d %d",numberOfThreads.x,numberOfThreads.y,numberOfThreads.z);
        
            float elapsed=0;
            hipEvent_t start, stop;

            hipEventCreate(&start);
            hipEventCreate(&stop);

            hipEventRecord(start, 0);

            for(int i=1;i<=100;i++)
               LANADP<<<numberOfBlocks,numberOfThreads>>>(d_arr,d_output,row,column);
        
            hipEventRecord(stop, 0);
            hipEventSynchronize (stop);

            hipEventElapsedTime(&elapsed, start, stop);
        
            printf("Time Elapsed: %f\n",elapsed);

            hipMemcpy(output,d_output,size_out,hipMemcpyDeviceToHost);
              

            // for(int i=0;i<row-2;i++) {
                
            //     for(int i1=0;i1<column-2;i1++) {
                    
            //         printf("%d  ",output[i][i1]);
            //     }
            //     printf("\n");
            // }

            std::ofstream outfile;

            outfile.open("lanadp_without_shared.txt", std::ios_base::app); // append instead of overwrite
            outfile <<argv[1]<<": " <<elapsed<<"\n"; 



        }
    }

    
 
   

   

  



}
