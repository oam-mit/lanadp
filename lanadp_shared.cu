#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "dcmtk/dcmimgle/dcmimage.h"

__constant__ int powers[9]={0,1,2,4,8,16,32,64,128};

#define NUMBER_OF_THREADS 16

__device__ int getGlobalID(int row,int col,int cols){
    return row*cols+col;
}

__device__ int getCircularIndex(int i){
    
    if (i<1) {
        return 8 + i;
    }
    else if (i>8 )
      return (i%9 )+1;
    else 
      return i;
  
}

//dichom, dcmtk
__global__ void LANADP (unsigned int *arr, unsigned char *out,unsigned int rows,unsigned int cols) {
    
    
    int global_row = threadIdx.y + blockDim.y * blockIdx.y;
    int global_col = threadIdx.x + blockDim.x * blockIdx.x;

    global_row++;
    global_col++;

    __shared__ short int shared_data[NUMBER_OF_THREADS+2][NUMBER_OF_THREADS+2];

    int local_row = threadIdx.y;
    int local_col = threadIdx.x;

    int shared_row = local_row+1;
    int shared_col = local_col+1;

    //printf("global_row: %d\nglobal_col:%d\nlocal_row:%d\nlocal_col:%d\nsahred_row: %d\nshared_col:%d\n",global_row,global_col,local_row,local_col,shared_row,shared_col);

    shared_data[shared_row][shared_col]=arr[getGlobalID(global_row,global_col,cols)];

    if (local_row==0){
        //if first_row, load data above itself compusorily
        shared_data[shared_row-1][shared_col]=arr[getGlobalID(global_row-1,global_col,cols)];

        //if first column, load diagonally above
        if(local_col==0){
            shared_data[shared_row-1][shared_col-1] = arr[getGlobalID(global_row-1,global_col-1,cols)];
        }

       //if last column load digonally above 
       if(local_col == gridDim.x-1){
            shared_data[shared_row-1][shared_col+1] = arr[getGlobalID(global_row-1,global_col+1,cols)];
        }
    }

    if (local_row==gridDim.y-1) {

        //if last row load elements below it
        shared_data[shared_row+1][shared_col] = arr[getGlobalID(global_row+1,global_col,cols)];

        //if first column, load diagonally below 
        if(local_col==0){
            shared_data[shared_row+1][shared_col-1] = arr[getGlobalID(global_row+1,global_col-1,cols)];
        }

        //if last column, load diagonally below
        if(local_col == gridDim.x-1){
            shared_data[shared_row+1][shared_col+1] = arr[getGlobalID(global_row+1,global_col+1,cols)];
        }
    }

    if (local_col==0){
        
        //if first column load elements to left
        shared_data[shared_row][shared_col-1] = arr[getGlobalID(global_row,global_col-1,cols)];
    
    }

    if(local_col==gridDim.x-1) {
        
        //if last column load elements to right
        shared_data[shared_row][shared_col+1] = arr[getGlobalID(global_row,global_col+1,cols)];
    }
	 
    __syncthreads();

  /*for (int i=0;i<3;i++){
      for(int i1=0;i1<3;i1++){
          printf("%d  ",shared_data[i][i1]);
      }
      printf("\n");
  }*/

 
       


    
    int mapping[9];
    mapping[1] = shared_data[shared_row][shared_col+1];
    mapping[2] = shared_data[shared_row-1][shared_col+1];
    mapping[3] = shared_data[shared_row-1][shared_col];
    mapping[4] = shared_data[shared_row-1][shared_col-1];
    mapping[5] = shared_data[shared_row][shared_col-1];
    mapping[6] = shared_data[shared_row+1][shared_col-1];
    mapping[7] = shared_data[shared_row+1][shared_col];
    mapping[8] = shared_data[shared_row+1][shared_col+1];

    /* mapping[1]=shared_arr[getGlobalID(global_row,global_col+1,cols)];
    mapping[2]=shared_arr[getGlobalID(global_row-1,global_col+1,cols)];
    mapping[3]=shared_arr[getGlobalID(global_row-1,global_col,cols)];
    mapping[4]=shared_arr[getGlobalID(global_row-1,global_col-1,cols)];
    mapping[5]=shared_arr[getGlobalID(global_row,global_col-1,cols)];
    mapping[6]=shared_arr[getGlobalID(global_row+1,global_col-1,cols)];
    mapping[7]=shared_arr[getGlobalID(global_row+1,global_col,cols)];
    mapping[8]=shared_arr[getGlobalID(global_row+1,global_col+1,cols)];*/


    int ans=0;
    for (int i=1;i<=8;i++) {
        
        float avg1 = ((float)(mapping[getCircularIndex(i+1)] + mapping[getCircularIndex(i+2)]))/2.0;
        float avg2 = ((float)(mapping[getCircularIndex(i-1)] + mapping[getCircularIndex(i-2)]))/2.0;

        //printf("Number: %d, Avg1: %f, Avg2: %f\n",mapping[i],avg1,avg2);

        if (avg1>=(float)mapping[i] && avg2>=(float)mapping[i] || (avg1<=(float)mapping[i] && avg2<=(float)mapping[i]) ){
            
            //printf("Adding:%d\n",powers[i]);
            ans+=powers[i];
        }
    }

    

    out[(global_row-1)*(cols-2)+(global_col-1)] = ans;


              
}


int main() {

    int row,column;
    printf("before readd\n");
    
    DicomImage *img=new DicomImage("512x512CT.dcm");
    

    if(img != NULL && img->getStatus()==EIS_Normal) {

        if(img->isMonochrome()) {

            img->setMinMaxWindow();
			
			row=img->getHeight();
			column=img->getWidth();

           

            unsigned int *arr = new unsigned int [row*column];

            printf("Rows: %d\tColumns:%d\n",row,column);
            printf("Image depth %d\n",img->getDepth());

            img->getOutputData(arr,row*column*sizeof(unsigned int));

            printf("after 4reading img\n");

            unsigned char output[row-2][column-2];

            int output_rows=row-2,output_colums=column-2;

            int size_arr = row*column*sizeof(unsigned int);
            int size_out = output_rows*output_colums*sizeof(unsigned char);

            //memory variables
            unsigned int *d_arr; 
            unsigned char *d_output;

            //allocating memory
            hipMalloc((void **)&d_arr,size_arr);
            hipMalloc((void **)&d_output,size_out);
        
            //copying data
            hipMemcpy(d_arr,arr,size_arr,hipMemcpyHostToDevice); 


            dim3 numberOfBlocks(ceil((float)(row-2)/16.0),ceil((float)(column-2)/16.0),1);
            dim3 numberOfThreads(NUMBER_OF_THREADS,NUMBER_OF_THREADS,1);
        
            float elapsed=0;
            hipEvent_t start, stop;

            hipEventCreate(&start);
            hipEventCreate(&stop);

            hipEventRecord(start, 0);

            for(int i=1;i<=50;i++)
                LANADP<<<numberOfBlocks,numberOfThreads>>>(d_arr,d_output,row,column);
        
            hipEventRecord(stop, 0);
            hipEventSynchronize (stop);

            hipEventElapsedTime(&elapsed, start, stop);
        
            printf("Time Elapsed: %f\n",elapsed);

            hipMemcpy(output,d_output,size_out,hipMemcpyDeviceToHost);
        
            
            // for(int i=0;i<row;i++)
            // {
            //     for(int i1=0;i1<column;i1++)
            //     {
            //         printf("%d ",output[i][i1]);
            //     }
            //     printf("\n");
            // }


        }
    }








    




}